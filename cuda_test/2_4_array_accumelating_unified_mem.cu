#include <stdio.h>
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"


#define NUM_EXECUTE 100

// Device code
__global__ void array_sum(int *d_a, int *d_b, int *d_c, int size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size) {
        d_c[gid] = d_a[gid] + d_b[gid];
    }
}

// Host code
int main() {
    std::cout << "Array accumalating CUDA programme" << std::endl;

    int size = 1920*1080*3;
    int block_size = 128;
    int NUM_BYTES = sizeof(int) * size;

    std::chrono::high_resolution_clock::time_point start_point;
    std::chrono::high_resolution_clock::time_point end_point;
    std::chrono::high_resolution_clock::time_point memcpy_point;
    std::chrono::duration<double, std::milli> delta;
    std::chrono::duration<double, std::milli> delta_memcpy;


    // Unified mem alloc
    int *a, *b, *c;
    hipMallocManaged((int **)&a, NUM_BYTES);
    hipMallocManaged((int **)&b, NUM_BYTES);
    hipMallocManaged((int **)&c, NUM_BYTES);

    // Prepare host data
    for (size_t i = 0; i < size; i++) {
        a[i] = 10;
        b[i] = 20;
    }
    memset(c, 0, NUM_BYTES);

    /* CPU version */
    for (size_t k = 0; k < NUM_EXECUTE; k++) {
        start_point = std::chrono::high_resolution_clock::now();
        for (size_t i = 0; i < size; i++) {
            c[i] = a[i] + b[i];
        }
        end_point = std::chrono::high_resolution_clock::now();  
        delta = (end_point - start_point);  
        // std::cout << "h_c[10]: " << h_c[10] << std::endl;
        std::cout << "Processing time CPU: " << delta.count() << " ms" << std::endl;
    }

    /* GPU version */
    dim3 block(block_size);
    dim3 grid(size/block_size);

    // Transfer data from host to device using pageable memory
    for (size_t k = 0; k < NUM_EXECUTE; k++) {
        start_point = std::chrono::high_resolution_clock::now();

        // cudaMemcpy(d_a, h_a, NUM_BYTES, cudaMemcpyHostToDevice);
        // cudaMemcpy(d_b, h_b, NUM_BYTES, cudaMemcpyHostToDevice);

        memcpy_point = std::chrono::high_resolution_clock::now();  

        // Kernel launch
        array_sum <<< grid, block >>> (a, b, c, NUM_BYTES);
        hipDeviceSynchronize();    

        end_point = std::chrono::high_resolution_clock::now();  
        delta = (end_point - start_point);  
        // std::cout << "c[10]: " << c[10] << std::endl;
        std::cout << "Processing time GPU: " << delta.count() << " ms" << std::endl;
        delta = (end_point - memcpy_point);  
        delta_memcpy = (memcpy_point - start_point);
        std::cout << "Memory transfer GPU: " << delta_memcpy.count() << " ms" << std::endl;
        std::cout << "Compute time GPU: " << delta.count() << " ms" << std::endl;
    }

    // Free memory
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}