#include <stdio.h>
#include <iostream>

#include "hip/hip_runtime.h"


// Device code
// tell compiler that following func is device code, not host
__global__
void cuda_kernel() { // return type is always void
    int row_offset = blockDim.x * blockDim.x * threadIdx.x;
    int col_offset = gridDim.x * blockIdx.y * blockDim.x * blockDim.y + blockDim.x * threadIdx.y;

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = tid + col_offset + row_offset;

    printf("threadId: %d, blockId: %d, globalId: %d\n", tid, bid, gid);
}

void device_prop() {
    int deviceNo = 0;
    hipDeviceProp_t iGpuProp;
    
    hipGetDeviceProperties(&iGpuProp, deviceNo);

    std::cout << "Max block size: ";
    for (size_t i = 0; i < 3; i++) {
        std::cout << iGpuProp.maxThreadsDim[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Max grid size: ";
    for (size_t i = 0; i < 3; i++) {
        std::cout << iGpuProp.maxGridSize[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Max thread per block: " << iGpuProp.maxThreadsPerBlock << std::endl;
}

// Host code
int main() {
    std::cout << "First CUDA programme" << std::endl;

    device_prop();

    // total threads in X, Y, Z dimension
    int nx, ny, nz;

    nx = 128;
    ny = nz = 1;

    // in 1 block: 32 threads in x, 1 thread in y & z
    dim3 block(32, 1, 1);

    // in 1 grid: 4 block in x, 1 block in y & z
    dim3 grid(nx/block.x, ny/block.y, nz/block.z);

    cuda_kernel <<< grid, block >>> (); 

    // host wait for device to execute
    hipDeviceSynchronize();
    
    // destroy CUDA context, device alloc ...
    hipDeviceReset();

    return 0;
}