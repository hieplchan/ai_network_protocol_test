#include <stdio.h>
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"


#define NUM_EXECUTE 100

// Device code
__global__ void array_sum(int *d_a, int *d_b, int *d_c, int size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size) {
        d_c[gid] = d_a[gid] + d_b[gid];
    }
}

// Host code
int main() {
    std::cout << "Array accumalating CUDA programme" << std::endl;

    int size = 1920*1080*3;
    int block_size = 128;
    int NUM_BYTES = sizeof(int) * size;

    std::chrono::high_resolution_clock::time_point start_point;
    std::chrono::high_resolution_clock::time_point end_point;
    std::chrono::high_resolution_clock::time_point memcpy_point;
    std::chrono::duration<double, std::milli> delta;
    std::chrono::duration<double, std::milli> delta_memcpy;


    // Host memmory alloc
    int *h_a, *h_b, *h_c;
    h_a = (int *)malloc(NUM_BYTES);
    h_b = (int *)malloc(NUM_BYTES);
    h_c = (int *)malloc(NUM_BYTES);

    // Prepare device memory
    int *d_a, *d_b, *d_c;
    hipMalloc((int **)&d_a, NUM_BYTES);
    hipMalloc((int **)&d_b, NUM_BYTES);
    hipMalloc((int **)&d_c, NUM_BYTES);

    // Prepare host data
    for (size_t i = 0; i < size; i++) {
        h_a[i] = 10;
        h_b[i] = 20;
    }
    memset(h_c, 0, NUM_BYTES);

    /* CPU version */
    for (size_t k = 0; k < NUM_EXECUTE; k++) {
        start_point = std::chrono::high_resolution_clock::now();
        for (size_t i = 0; i < size; i++) {
            h_c[i] = h_a[i] + h_b[i];
        }
        end_point = std::chrono::high_resolution_clock::now();  
        delta = (end_point - start_point);  
        // std::cout << "h_c[10]: " << h_c[10] << std::endl;
        std::cout << "Processing time CPU: " << delta.count() << " ms" << std::endl;
    }

    /* GPU version */
    memset(h_c, 0, NUM_BYTES);
    dim3 block(block_size);
    dim3 grid(size/block_size);

    // Transfer data from host to device using pageable memory
    for (size_t k = 0; k < NUM_EXECUTE; k++) {
        start_point = std::chrono::high_resolution_clock::now();

        hipMemcpy(d_a, h_a, NUM_BYTES, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, NUM_BYTES, hipMemcpyHostToDevice);

        memcpy_point = std::chrono::high_resolution_clock::now();  

        // Kernel launch
        array_sum <<< grid, block >>> (d_a, d_b, d_c, NUM_BYTES);
        hipDeviceSynchronize();    
        hipMemcpy(h_c, d_c, NUM_BYTES, hipMemcpyDeviceToHost);

        end_point = std::chrono::high_resolution_clock::now();  
        delta = (end_point - start_point);  
        // std::cout << "h_c[10]: " << h_c[10] << std::endl;
        std::cout << "Processing time GPU: " << delta.count() << " ms" << std::endl;
        delta = (end_point - memcpy_point);  
        delta_memcpy = (memcpy_point - start_point);
        std::cout << "Memory transfer GPU: " << delta_memcpy.count() << " ms" << std::endl;
        std::cout << "Compute time GPU: " << delta.count() << " ms" << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}